#include "hip/hip_runtime.h"
#include <>
#include "taichi/ui/backends/vulkan/vertex.h"

namespace taichi {
namespace ui {

int div_up(int a, int b) {
  if (b == 0) {
    return 1;
  }
  int result = (a % b != 0) ? (a / b + 1) : (a / b);
  return result;
}

#define MAX_THREADS_PER_BLOCK 1024
void set_num_blocks_threads(int N, int &num_blocks, int &num_threads) {
  num_threads = min(N, MAX_THREADS_PER_BLOCK);
  num_blocks = div_up(N, num_threads);
}

__global__ void update_renderables_vertices_cuda_impl(Vertex *vbo,
                                                      float *vertices,
                                                      int num_vertices,
                                                      int num_components) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_vertices)
    return;

  vbo[i].pos.x = vertices[i * num_components];
  vbo[i].pos.y = vertices[i * num_components + 1];
  if (num_components == 3) {
    vbo[i].pos.z = vertices[i * num_components + 2];
  }
}

void update_renderables_vertices_cuda(Vertex *vbo,
                                      float *vertices,
                                      int num_vertices,
                                      int num_components) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_vertices, num_blocks, num_threads);
  update_renderables_vertices_cuda_impl<<<num_blocks, num_threads>>>(
      vbo, vertices, num_vertices, num_components);
}

void update_renderables_vertices_x64(Vertex *vbo,
                                     float *vertices,
                                     int num_vertices,
                                     int num_components) {
  for (int i = 0; i < num_vertices; ++i) {
    vbo[i].pos.x = vertices[i * num_components];
    vbo[i].pos.y = vertices[i * num_components + 1];
    if (num_components == 3) {
      vbo[i].pos.z = vertices[i * num_components + 2];
    }
  }
}

__global__ void update_renderables_indices_cuda_impl(int *ibo,
                                                     int *indices,
                                                     int num_indices) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_indices)
    return;

  ibo[i] = indices[i];
}
void update_renderables_indices_cuda(int *ibo, int *indices, int num_indices) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_indices, num_blocks, num_threads);
  update_renderables_indices_cuda_impl<<<num_blocks, num_threads>>>(
      ibo, indices, num_indices);
}

void update_renderables_indices_x64(int *ibo, int *indices, int num_indices) {
  for (int i = 0; i < num_indices; ++i) {
    ibo[i] = indices[i];
  }
}

__global__ void update_renderables_colors_cuda_impl(Vertex *vbo,
                                                    float *colors,
                                                    int num_vertices) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_vertices)
    return;

  vbo[i].color.x = colors[i * 3];
  vbo[i].color.y = colors[i * 3 + 1];
  vbo[i].color.z = colors[i * 3 + 2];
}
void update_renderables_colors_cuda(Vertex *vbo,
                                    float *colors,
                                    int num_vertices) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_vertices, num_blocks, num_threads);
  update_renderables_colors_cuda_impl<<<num_blocks, num_threads>>>(
      vbo, colors, num_vertices);
}

void update_renderables_colors_x64(Vertex *vbo,
                                   float *colors,
                                   int num_vertices) {
  for (int i = 0; i < num_vertices; ++i) {
    vbo[i].color.x = colors[i * 3];
    vbo[i].color.y = colors[i * 3 + 1];
    vbo[i].color.z = colors[i * 3 + 2];
  }
}

__global__ void update_renderables_normals_cuda_impl(Vertex *vbo,
                                                     float *normals,
                                                     int num_vertices) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_vertices)
    return;

  vbo[i].normal.x = normals[i * 3];
  vbo[i].normal.y = normals[i * 3 + 1];
  vbo[i].normal.z = normals[i * 3 + 2];
}
void update_renderables_normals_cuda(Vertex *vbo,
                                     float *normals,
                                     int num_vertices) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_vertices, num_blocks, num_threads);
  update_renderables_normals_cuda_impl<<<num_blocks, num_threads>>>(
      vbo, normals, num_vertices);
}
void update_renderables_normals_x64(Vertex *vbo,
                                    float *normals,
                                    int num_vertices) {
  for (int i = 0; i < num_vertices; ++i) {
    vbo[i].normal.x = normals[i * 3];
    vbo[i].normal.y = normals[i * 3 + 1];
    vbo[i].normal.z = normals[i * 3 + 2];
  }
}

template <typename T>
__device__ __host__ inline unsigned char get_color_value(T x);

template <>
__device__ __host__ inline unsigned char get_color_value<unsigned char>(
    unsigned char x) {
  return x;
}

template <>
__device__ __host__ inline unsigned char get_color_value<float>(float x) {
  x = max(0.f, min(1.f, x));
  return (unsigned char)(x * 255);
}

template <typename T>
__global__ void copy_to_texture_fuffer_cuda_impl(T *src,
                                                 uint64_t surface,
                                                 int width,
                                                 int height,
                                                 int actual_width,
                                                 int actual_height,
                                                 int channels) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= width * height)
    return;

  int y = i / width;
  int x = i % width;

  T *src_base_addr = src + (x * actual_height + y) * channels;
  uchar4 data = make_uchar4(0, 0, 0, 0);

  data.x = get_color_value<T>(src_base_addr[0]);
  data.y = get_color_value<T>(src_base_addr[1]);
  data.z = get_color_value<T>(src_base_addr[2]);
  data.w = 255;

  surf3Dwrite(data, surface, x * sizeof(uchar4), y, 0);
}

template <typename T>
void copy_to_texture_fuffer_cuda(T *src,
                                 uint64_t surface,
                                 int width,
                                 int height,
                                 int actual_width,
                                 int actual_height,
                                 int channels) {
  int num_blocks, num_threads;
  set_num_blocks_threads(width * height, num_blocks, num_threads);
  copy_to_texture_fuffer_cuda_impl<<<num_blocks, num_threads>>>(
      src, (uint64_t)surface, width, height, actual_width, actual_height,
      channels);
}

template <typename T>
void copy_to_texture_fuffer_x64(T *src,
                                unsigned char *dest,
                                int width,
                                int height,
                                int actual_width,
                                int actual_height,
                                int channels) {
  for (int i = 0; i < width * height; ++i) {
    int y = i / width;
    int x = i % width;

    T *src_base_addr = src + (x * actual_height + y) * channels;
    uchar4 data = make_uchar4(0, 0, 0, 0);

    data.x = get_color_value<T>(src_base_addr[0]);
    data.y = get_color_value<T>(src_base_addr[1]);
    data.z = get_color_value<T>(src_base_addr[2]);
    data.w = 255;

    ((uchar4 *)dest)[y * width + x] = data;
  }
}

template void copy_to_texture_fuffer_cuda<float>(float *src,
                                                 uint64_t surface,
                                                 int width,
                                                 int height,
                                                 int actual_width,
                                                 int actual_height,
                                                 int channels);
template void copy_to_texture_fuffer_cuda<unsigned char>(unsigned char *src,
                                                         uint64_t surface,
                                                         int width,
                                                         int height,
                                                         int actual_width,
                                                         int actual_height,
                                                         int channels);

template void copy_to_texture_fuffer_x64<float>(float *src,
                                                unsigned char *dest,
                                                int width,
                                                int height,
                                                int actual_width,
                                                int actual_height,
                                                int channels);
template void copy_to_texture_fuffer_x64<unsigned char>(unsigned char *src,
                                                        unsigned char *dest,
                                                        int width,
                                                        int height,
                                                        int actual_width,
                                                        int actual_height,
                                                        int channels);

}  // namespace ui
}  // namespace taichi
